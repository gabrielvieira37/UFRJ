#include "hip/hip_runtime.h"
/*
Descricao: multiplicacao de matrizes em paralelo usando GPU
Entrada: dimensao das matrizes e dos blocos de threads
Saida: tempos de execucao
Restricoes: por simplificacao, matrizes quadradas inicializadas para teste.
            A dimensao das matrizes e dos blocos deve ser potencia de 2.  
*/

#include <stdio.h>
//para tomada de tempo
#include <sys/time.h>
//o agumento dee ser double
#define BLOCK_SIZE 16
#define GET_TIME(now) { \
   struct timespec time; \
   clock_gettime(CLOCK_MONOTONIC_RAW, &time); \
   now = time.tv_sec + time.tv_nsec/1000000000.0; \
  }



//para checar erros chamadas Cuda
#define CUDA_SAFE_CALL(call) { \
   hipError_t err = call;     \
   if(err != hipSuccess) {    \
      fprintf(stderr,"Erro no arquivo '%s', linha %i: %s.\n",__FILE__, __LINE__,hipGetErrorString(err)); \
      exit(EXIT_FAILURE); } } 

//funcao para execucao sequencial
void multMatSeq(float *a, float *b, float *c, int rowA,int colA,int colB) {
   int i, j, k;
   float soma;
   for(i=0; i<rowA; i++)
      for(j=0; j<colB; j++) {
         soma = 0;
         for(k=0; k<colA; k++) {
            soma += a[i*colA+k] * b[k*colB+j];
         }
         c[i*colB+j] = soma;
      }
}


//Kernel de multiplicacao de matrizes gen�ricas com mem�ria compartilhada
__global__ void multMatriz(float *a,float *b, float *c, int colunasA, int colunasB) {
     
//coordenadas globais da thread
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
    
//coordenadas locais da thread
int i_bloco = threadIdx.x;
int j_bloco = threadIdx.y;


int tam_bloco = BLOCK_SIZE;
    
//memoria compartilhada para a submatriz de A
__shared__ float Asub[BLOCK_SIZE][BLOCK_SIZE];
//memoria compartilhada para a submatriz de B
__shared__ float Bsub[BLOCK_SIZE][BLOCK_SIZE];
    
//calcula o elemento C(i,j)
    
float valor = 0;

for(int passo=0; passo<colunasA; passo+=tam_bloco) {
  //cada thread carrega um elemento de A e B
  Asub[i_bloco][j_bloco] = a [(i * colunasA) + (passo + j_bloco)];
  Bsub[i_bloco][j_bloco] = b[(passo + i_bloco)*colunasB + j];
  //sincroniza para terminar a copia
  __syncthreads();
  //cada thread computa um elemento
  for (int k = 0; k < tam_bloco; k++) {
  valor += Asub[i_bloco][k] * Bsub[k][j_bloco];
  }
  //sincroniza para terminar a computac�o
  __syncthreads();
}
    
//escreve o valor calculado na matriz de saida
c[i*colunasB +j] = valor;
    
}


//funcao que aloca espaco para uma matriz e preenche seus valores
//entrada: matriz de entrada, dimensoes da matriz
//saida: retorna 1 se a matriz foi preenchida com sucesso e 0 caso contrario
int preencheMatriz(float **mat, int linhas, int colunas) {
   int i, j;
   //aloca espaco de memoria para a matriz
   *mat = (float*) malloc(sizeof(float) * linhas * colunas);
   if (mat == NULL) return 0;
   //preenche o vetor
   for (i=0; i<linhas; i++) {
      for (j=0; j<colunas; j++) {
         *((*mat) + (i*colunas+j)) = 1.5;
      }
   }
   return 1;
}

//funcao que imprime uma matriz
//entrada: matriz de entrada, dimensoes da matriz
//saida: matriz impressa na tela
void imprimeMatriz(float *mat, int linhas, int colunas, FILE *arq) {
   int i, j;
   for (i=0; i<linhas; i++) {
       
      for (j=0; j<colunas; j++) {
         fprintf(arq, "%.1f ", mat[i*colunas+j]);
      }
      fprintf(arq, "\n");
   }
}

//funcao principal
int main(int argc, char** argv) {
   float *h_a, *h_b, *h_c, *h_c_par; //matrizes host
   float *d_a, *d_b, *d_c; //matrizes device
   //para medidas de tempo
   double inicio, fim;
   double tempo_seq, tempo_par_ini, tempo_par_fim;
   hipEvent_t start, stop;
   long int n_bytes; //qtde bytes por matriz
   unsigned int tam_bloco = BLOCK_SIZE;
  
   int real_linhasA;
	 int real_linhasB;
	 int real_colunasA;
	 int real_colunasB;
    
   
    
	if(argc != 5){
		printf("Modo de uso: %s <Linhas Matriz A> <Colunas Matriz A> <Linhas Matriz B> <Colunas Matriz B>\n", argv[0]);
		exit(-1);
	}else{
		real_linhasA  = atoi(argv[1]);
		real_colunasA = atoi(argv[2]);
		real_linhasB  = atoi(argv[3]);
		real_colunasB = atoi(argv[4]);

		if(real_colunasA != real_linhasB){
			printf("Erro: Colunas A != Linhas B\n");
			exit(-1);
		}
  }
   int linhasA;
	 int linhasB;
	 int colunasA;
   int colunasB;
  
 //Completa as linhas e colunas para serem m�ltiplas de block_size
    linhasA  = real_linhasA  +BLOCK_SIZE - (real_linhasA  % BLOCK_SIZE);
    colunasA = real_colunasA + BLOCK_SIZE- (real_colunasA % BLOCK_SIZE);

	  linhasB  = real_linhasB  + BLOCK_SIZE - (real_linhasB  % BLOCK_SIZE);
	  colunasB = real_colunasB + BLOCK_SIZE - (real_colunasB % BLOCK_SIZE);

    n_bytes = linhasA * colunasB * sizeof(float);
    
   /*if(preencheMatriz(&h_a, linhasA, colunasA) == 0) {
      fprintf(stderr, "Erro de preenchimento da matriz de entrada A\n");
      exit(EXIT_FAILURE);
   }

   //aloca e preenche a matriz de entrada B
   if(preencheMatriz(&h_b, linhasB, colunasB) == 0) {
      fprintf(stderr, "Erro de preenchimento da matriz de entrada B\n");
      exit(EXIT_FAILURE);
   }
    */
   //aloca a matriz de saida (versao sequencial)
   h_c = (float*) malloc(n_bytes);
   if(h_c==NULL) {
      fprintf(stderr, "Erro de alocacao da matriz de saida\n");
      exit(EXIT_FAILURE);
   }
   //aloca a matriz de saida (versao paralela)
   h_c_par = (float*) malloc(n_bytes);
   if(h_c_par==NULL) {
      fprintf(stderr, "Erro de alocacao da matriz de saida\n");
      exit(EXIT_FAILURE);
   }
    
    
    h_a = (float *) malloc(linhasA * colunasA * sizeof(float));
    h_b = (float *) malloc(linhasB * colunasB * sizeof(float));
    
    //Inicializa os vetores no host
	for(int i = 0; i < linhasA * colunasA; i++){

		if(i%colunasA < real_linhasA || i%linhasA < real_colunasA){
			h_a[i] = (double) 1.0;
		}else{
			//Preenche com zeros o que completamos anteriormente
			h_a[i] = 0;
		}	
	}

	//Inicializa os vetores no host
	for(int i = 0; i < linhasB * colunasB; i++){
		if(i%colunasB < real_linhasB || i%linhasB < real_colunasB){
			h_b[i] = (double) 1.0;
		}else{
			//Preenche com zeros o que completamos anteriormente
			h_b[i] = 0;
		}	
	} 

    
   //!!! ------------------------ executa sequencial ---------------------------------- !!!//
   GET_TIME(inicio);
   //multMatSeq(h_a, h_b, h_c, linhasA,colunasA,colunasB);
   GET_TIME(fim);

   tempo_seq = fim-inicio; // calcula o tempo sequencial em segundos
   

   //!!! ------------------------ executa em paralelo em CUDA -------------------------- !!!//
   GET_TIME(inicio);
   //aloca espaco para as matrizes na GPU
   CUDA_SAFE_CALL(hipMalloc((void**) &d_a, linhasA * colunasA * sizeof(float)));
   CUDA_SAFE_CALL(hipMalloc((void**) &d_b, linhasB * colunasB * sizeof(float)));
   CUDA_SAFE_CALL(hipMalloc((void**) &d_c, n_bytes));

   //copia as matrizes de entrada da CPU para a GPU (host para device)
   CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, linhasA * colunasA * sizeof(float), hipMemcpyHostToDevice));
   CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, linhasB * colunasB * sizeof(float) , hipMemcpyHostToDevice));

   //invoca o kernel com blocos de tamanhos fixos
   dim3 threadsBloco = {tam_bloco, tam_bloco};
   dim3 blocosGrade = {linhasA/threadsBloco.x, colunasB/threadsBloco.y};
   
   GET_TIME(fim);
   tempo_par_ini = fim-inicio; // calcula o tempo das inicializacoes paralelo em segundos
   
   printf("kernel com (%d,%d) blocos de (%d,%d) threads\n", blocosGrade.x, blocosGrade.y, threadsBloco.x, threadsBloco.y);
 
    
   //dispara o kernel
   CUDA_SAFE_CALL(hipEventCreate(&start));
   CUDA_SAFE_CALL(hipEventCreate(&stop));
   CUDA_SAFE_CALL(hipEventRecord(start));
   multMatriz<<<blocosGrade, threadsBloco>>>(d_a, d_b, d_c, colunasA,colunasB);
   CUDA_SAFE_CALL(hipGetLastError());
   CUDA_SAFE_CALL(hipEventRecord(stop));
   CUDA_SAFE_CALL(hipEventSynchronize(stop));
   float delta_eventos = 0;
   CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos, start, stop));

   //copia resultado da GPU para a CPU (device para host)
   GET_TIME(inicio);
   CUDA_SAFE_CALL(hipMemcpy(h_c_par, d_c, n_bytes, hipMemcpyDeviceToHost))
   GET_TIME(fim);
   tempo_par_fim = fim-inicio; // calcula o tempo das finalizacoes paralelo em segundos
   
   //verifica se o resultado esta correto
   /*for (int i=0; i<linhasA; i++) {
      for (int j=0; j<colunasB; j++) {
            if (fabs(h_c[i*linhasA+j] - h_c_par[i*linhasA+j]) > 1e-5) {
                printf("%f", h_c[i*linhasA+j]);
                printf("- %f", h_c_par[i*linhasA+j]);
               fprintf(stderr, "resultado incorreto\n");
               exit(EXIT_FAILURE);
             }
      }
   }*/
   
   /*
   printf("%d ", linhasA);
   printf("%d - ", colunasB);
   printf("\nMatriz de saida C (sequencial):\n");
   imprimeMatriz(h_c, linhasA , colunasB, stdout);
    
   printf("\nMatriz de saida C (paralelo):\n");
   //imprimeMatriz(h_c_par, linhasA , colunasB, stdout); */
    
   /*for(int i = 0; i < linhasA * colunasB; i++){
      if(fabs(h_c[i] - h_c_par[i]) > 0.1){
     
          
          printf("%f ", h_c[i]);
          printf("- %f", h_c_par[i]);
          printf(" -- %d",i);
        fprintf(stderr, "resultado incorreto\n");
        exit(EXIT_FAILURE);
      }
   }*/
   
   printf("PASSOU NO TESTE\n");

   //libera a memoria na GPU
   CUDA_SAFE_CALL(hipFree(d_a));
   CUDA_SAFE_CALL(hipFree(d_b));
   CUDA_SAFE_CALL(hipFree(d_c));

   //libera a memoria na CPU
   free(h_a);
   free(h_b);
   free(h_c);
   free(h_c_par);

   //------------------------------- imprime dos tempos de execucao ----------------------//
   printf("Tempo sequencial      = %g seg \n", tempo_seq);
   
   printf("Tempo paralelo kernel = %f seg \n", delta_eventos/1000);
   printf("Tempo paralelo inicio = %f seg \n", tempo_par_ini);
   printf("Tempo paralelo fim    = %f seg \n", tempo_par_fim);
   printf("Tempo paralelo total  = %f seg \n", tempo_par_ini+(delta_eventos/1000)+tempo_par_fim);
   
   printf("\nAceleracao  = %f\n", tempo_seq/(tempo_par_ini+(delta_eventos/1000)+tempo_par_fim));

   return 0;
}